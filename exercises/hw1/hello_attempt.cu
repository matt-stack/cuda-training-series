
#include <hip/hip_runtime.h>
#include <stdio.h>

#define N 2
#define M 2

__global__ void hello(){

  printf("Hello from block: %d, thread: %d\n", blockIdx.x, threadIdx.x);
}

int main(){

  hello<<< N, M >>>();
  hipDeviceSynchronize();
}

