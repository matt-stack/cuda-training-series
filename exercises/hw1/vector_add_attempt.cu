#define N 2048*2048
#define M 512


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

__global__ void vector_add(int *d_a, int *d_b, int *d_c, int n){
	int index = threadIdx.x + blockIdx.x * blockDim.x;
	if (index < n){
		d_c[index] = d_a[index] + d_b[index];	
	}
}

int main(){
	int *a, *b, *c; // host
	int *d_a, *d_b, *d_c; // device
	int size = sizeof(int) * N;
	srand (time(NULL));

	//alloc device copies
	hipMalloc((void **) &d_a, size);
	hipMalloc((void **) &d_b, size);
	hipMalloc((void **) &d_c, size);

	//alloc host copies
	a = (int *)malloc(size);
	b = (int *)malloc(size);
	c = (int *)malloc(size);

	for (int i = 0; i < N; i ++){
		a[i] = rand() % 100;
		b[i] = rand() % 100;
	}

	hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
	hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

	vector_add<<<(N + M-1)/M, M>>>(d_a, d_b, d_c, N);

	hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);

	//for (int i = 0; i < N; i ++){
		printf("a[0] = %d \n b[0] = %d \n c[0] = %d \n", a[0], b[0], c[0]);
	//}
	

	free(a);
	free(b);
	free(c);
	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);
}
